#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

struct Matrix{
int width;
int height;
float *elements;
};

extern “C” device float getElement(Matrix *A, int row, int col){
return A->elements[row * A->width + col];
}

extern “C” device void setElement(Matrix *A, int row, int col, float value){
A->elements[row * A->width + col] = value;
}

extern “C” global void matMulKernel(Matrix *A, Matrix *B, Matrix *C){
float Cvalue = 0.0;
int row = threadIdx.y + blockIdx.y * blockDim.y;
int col = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = 0; i < A->width; ++i){
            Cvalue += getElement(A, row, i) * getElement(B, i, col);
    }
    setElement(C, row, col, Cvalue);
}

extern “C” int matrixX(float *h_A,float *h_B,float *h_C_host,int w){
int width = w;
int height = w;
double timeuse;

Matrix *A, *B, *C;
hipMallocManaged((void**)&A, sizeof(Matrix));
hipMallocManaged((void**)&B, sizeof(Matrix));
hipMallocManaged((void**)&C, sizeof(Matrix));
 
int nBytes = width * height * sizeof(float);
 
hipMallocManaged((void**)&A->elements, nBytes);
hipMallocManaged((void**)&B->elements, nBytes);
hipMallocManaged((void**)&C->elements, nBytes);
 
A->height = height;
A->width = width;
B->height = height;
B->width = width;
C->height = height;
C->width = width;

hipMemcpy(A->elements,h_A,w*w*sizeof(float),hipMemcpyHostToDevice);
hipMemcpy(B->elements,h_B,w*w*sizeof(float),hipMemcpyHostToDevice);
 
dim3 blockSize(16, 16);
dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
    (height + blockSize.y - 1) / blockSize.y);
 
struct timeval t1,t2;
gettimeofday(&t1,NULL);
 
matMulKernel << < gridSize, blockSize >> >(A, B, C);
 
 hipDeviceSynchronize();//After the kernel function call, hipDeviceSynchronize() must be called to wait for the device to complete the access before the CPU can access it
 
hipMemcpy(h_C_host,C->elements,w*w*sizeof(float),hipMemcpyDeviceToHost);
 
gettimeofday(&t2,NULL);
timeuse = t2.tv_sec - t1.tv_sec + (t2.tv_usec - t1.tv_usec)/1000000.0;
printf("Use Time:%fs\n", timeuse);
hipFree(A->elements);
hipFree(B->elements);
hipFree(C->elements);
hipFree(A);
hipFree(B);
hipFree(C);
 
return 0;
}